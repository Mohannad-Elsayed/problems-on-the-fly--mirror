#include <iostream>
#include <vector>
#include <numeric> // For gcd
#include <algorithm> // For lcm
#include <hip/hip_runtime.h>

// Custom atomicAdd for long long
__device__ long long atomicAddLongLong(long long* address, long long val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, val + assumed);
    } while (assumed != old);
    return old;
}

using namespace std;
using ll = long long;
const int MOD = 1000000007; // Fixed 1e9 + 7 to explicit integer value

// Helper function to compute gcd (needed for device code)
__device__ __host__ ll gcd(ll a, ll b) {
    while (b != 0) {
        ll temp = b;
        b = a % b;
        a = temp;
    }
    return a;
}

// CUDA kernel to compute gcd and lcm in parallel
__global__ void computeLcmKernel(int* v, int n, ll* result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < n && j < n && i < j) {
        ll gcd_val = gcd(v[i], v[j]);
        ll lcm_val = (v[i] / gcd_val) * v[j];
        atomicAddLongLong(result, lcm_val % MOD);
        // atomicAdd(result, lcm_val % MOD);
    }
}

// Host function
void stress() {
    int n, k;
    cin >> n >> k;
    vector<int> v(n);
    for (int i = 0; i < n; i++) {
        cin >> v[i];
    }

    // Allocate memory on the device
    int* d_v;
    ll* d_result;
    ll h_result = 0;

    hipMalloc(&d_v, n * sizeof(int));
    hipMalloc(&d_result, sizeof(ll));
    hipMemcpy(d_v, v.data(), n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_result, &h_result, sizeof(ll), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((n + blockDim.x - 1) / blockDim.x, (n + blockDim.y - 1) / blockDim.y);

    // Launch the kernel
    computeLcmKernel<<<gridDim, blockDim>>>(d_v, n, d_result);

    // Copy the result back to the host
    hipMemcpy(&h_result, d_result, sizeof(ll), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_v);
    hipFree(d_result);

    // Output the result
    cout << h_result % MOD << endl;
}

// Added main function to call stress
int main() {
    stress();
    return 0;
}