#include <hip/hip_runtime.h>

#include <fstream>  // For ofstream
#include <algorithm> // For min
#include <iostream>
#include <vector>

using ll = long long;

// Device function for GCD calculation
__device__ int device_gcd(int a, int b) {
    while (b) {
        int temp = b;
        b = a % b;
        a = temp;
    }
    return a;
}

// Device function for LCM calculation
__device__ int device_lcm(int a, int b) {
    int gcd = device_gcd(a, b);
    return (a / gcd) * b; // Safe division first to prevent overflow
}

// Device function to check if number is prime
__device__ bool device_isPrime(long long n) {
    if (n <= 1)
        return false;
    
    int primes[] = {2, 3, 5, 7, 11, 13, 17, 19, 23, 29};
    for (int i = 0; i < 10; i++) {
        if (n == primes[i])
            return true;
        if (n % primes[i] == 0)
            return false;
    }
    
    int arr[] = {1, 7, 11, 13, 17, 19, 23, 29};
    // 30k + 1, 30k + 7
    for (long long i = 30; i * i <= n; i += 30)
        for (int j = 0; j < 8; j++)
            if (n % (i+arr[j]) == 0)
                return false;

    return true;
}

// Host function to check if number is prime
bool isPrime(ll n) {
    if (n <= 1)
        return false;
    
    int primes[] = {2, 3, 5, 7, 11, 13, 17, 19, 23, 29};
    int primesc = sizeof(primes) / sizeof(primes[0]);
    for (int i = 0; i<primesc; i++) {
        if (n == primes[i])
            return true;
        if (n % primes[i] == 0)
            return false;
    }
    
    int arr[] = {1, 7, 11, 13, 17, 19, 23, 29};
    int arrc = sizeof(arr)/sizeof(arr[0]);
    // 30k + 1, 30k + 7
    for (ll i = 30; i * i <= n; i += 30)
        for (int j = 0; j < arrc; j++)
            if (n % (i+arr[j]) == 0)
                return false;

    return true;
}

// CUDA kernel for computing prime check on LCM/GCD
__global__ void countPrimesKernel(int n, int* count) {
    int a = blockIdx.x + 1;
    int b = threadIdx.x + a + 1;
    
    if (a <= n && b <= n) {
        int g = device_gcd(a, b);
        int l = (a / g) * b; // Calculate LCM more safely
        if (device_isPrime(l / g)) {
            atomicAdd(count, 1);
        }
    }
}

// Helper function to check CUDA errors
void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s (%s)\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void stress() {
    const int MAX_N = 10000; // Reduced for testing
    std::vector<int> res(MAX_N + 1);
    int* d_count;
    
    hipError_t err = hipMalloc(&d_count, sizeof(int));
    checkCudaError(err, "Failed to allocate device memory");
    
    for (int n = 1; n <= MAX_N; n++) {
        // Reset counter to 0
        int host_count = 0;
        err = hipMemcpy(d_count, &host_count, sizeof(int), hipMemcpyHostToDevice);
        checkCudaError(err, "Failed to copy count to device");
        
        // Launch kernel with appropriate dimensions
        int threadsPerBlock = std::min(n, 1024);
        int numBlocks = n;
        countPrimesKernel<<<numBlocks, threadsPerBlock>>>(n, d_count);
        
        // Check for kernel launch errors
        err = hipGetLastError();
        checkCudaError(err, "Kernel launch failed");
        
        // Synchronize before copying results back
        err = hipDeviceSynchronize();
        checkCudaError(err, "Kernel execution failed");
        
        // Copy result back
        err = hipMemcpy(&host_count, d_count, sizeof(int), hipMemcpyDeviceToHost);
        checkCudaError(err, "Failed to copy count from device");
        
        res[n] = host_count;
        
        if (n % 1000 == 0) {
            std::cout << "Completed n = " << n << ", count = " << host_count << std::endl;
        }
    }
    
    hipFree(d_count);
    
    // Print or save results
    std::ofstream outfile("results.txt");
    if (!outfile.is_open()) {
        std::cerr << "Failed to open output file" << std::endl;
        return;
    }
    for (int i = 1; i <= MAX_N; i++) {
        outfile << i << " " << res[i] << std::endl;
    }
    outfile.close();
}

int main() {
    // Initialize CUDA
    hipError_t err = hipSetDevice(0);
    checkCudaError(err, "Failed to set CUDA device");
    
    stress();
    return 0;
}