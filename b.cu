#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

using ll = long long;
#define int ll
const int MOD = (int)1e9 + 7;

//---------------------------------------------------------
// Device function: count1
//---------------------------------------------------------
__device__ ll d_count1(ll X, int k) {
    if (X < 0) return 0;
    ll cycle = 1LL << (k + 1);
    ll half = 1LL << k;
    ll full = (X + 1) / cycle;
    ll res = full * half;
    ll rem = (X + 1) % cycle;
    res += (rem > half ? rem - half : 0);
    return res;
}

//---------------------------------------------------------
// Kernel for optimized o(n) (computed by a single thread)
//---------------------------------------------------------
__global__ void o_kernel(int n, int *d_result) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        ll ans = 0;
        ll ones_cache[64];
        for (int i = 0; i < 64; i++)
            ones_cache[i] = -1;
        for (int k = 0; (1LL << k) <= n; k++) {
            ll p = 1LL << k;
            ll totalCount = (n - p + 1);
            ll ones = (ones_cache[k] == -1 ? d_count1(n - p, k) : ones_cache[k]);
            ones_cache[k] = ones;
            ll count0 = totalCount - ones;
            ll onesAll = d_count1(n, k);
            ans = (ans + count0 + onesAll) % MOD;
        }
        *d_result = ans % MOD;
    }
}

//---------------------------------------------------------
// Kernel for brute-force b(n)
// Each thread computes one (i,j) pair.
//---------------------------------------------------------
__global__ void b_kernel(int n, int *d_result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int local = 0;
    if (i <= n && j <= n) {
        if (__popc(i ^ j) == 1) {
            local = 1;
        }
    }
    // Fix: Use atomicAdd with `unsigned long long`
    if (local)
        atomicAdd((unsigned long long*)d_result, (unsigned long long)local);
}

//---------------------------------------------------------
// Host wrapper: Compute b(n) on GPU
//---------------------------------------------------------
int b_gpu(int n) {
    int h_result = 0;
    int *d_result;
    hipMalloc(&d_result, sizeof(int));
    hipMemcpy(d_result, &h_result, sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((n + threadsPerBlock.x) / threadsPerBlock.x,
                   (n + threadsPerBlock.y) / threadsPerBlock.y);
    b_kernel<<<numBlocks, threadsPerBlock>>>(n, d_result);
    hipDeviceSynchronize();
    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_result);
    return h_result % MOD;
}

//---------------------------------------------------------
// Host wrapper: Compute o(n) on GPU
//---------------------------------------------------------
int o_gpu(int n) {
    int h_result = 0;
    int *d_result;
    hipMalloc(&d_result, sizeof(int));
    hipMemcpy(d_result, &h_result, sizeof(int), hipMemcpyHostToDevice);

    o_kernel<<<1, 1>>>(n, d_result);
    hipDeviceSynchronize();
    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_result);
    return h_result % MOD;
}

//---------------------------------------------------------
// Main: test for n from 0 to 9999
//---------------------------------------------------------
signed main() {
    // for (int n = 0; n <= 1000000; n++) {
    //     int res_o = o_gpu(n);
    //     int res_b = b_gpu(n);
    //     if (res_o != res_b) {
    //         cout << "Mismatch at n = " << n << ": o(n) = " << res_o << " vs b(n) = " << res_b << "\n";
    //         return 0;
    //     }
    //     cout << n << ' ';
    // }
    // cout << "\nAll Passed!";
    // cout << b_gpu(1000000);
    return 0;
}
